
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

__global__
void flops(int n, float c, float *x, float *y){

  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if(c==2.0f){
    if (i < n) y[i] = c*x[i] + y[i];
  }else{
    if (i<n){};
  }

}

__global__
void iops(int n, int c, int *a, int *b){

  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) b[i] = c*a[i] + b[i];

}

double read_timer(){

    struct timeval start;
    gettimeofday( &start, NULL );
    return (double)((start.tv_sec) + 1.0e-6 * (start.tv_usec))*1000;

}

int main(void){

  printf("\n\nYou are executing the GPU Benchmarking\n\n");
   
  int N = 20*(1<<20);
  float *x, *y, *dev_x, *dev_y;
  int *a, *b, *dev_a, *dev_b;

  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  a = (int*)malloc(N*sizeof(int));
  b = (int*)malloc(N*sizeof(int));

  hipMalloc(&dev_x, N*sizeof(float));
  hipMalloc(&dev_y, N*sizeof(float));

  hipMalloc(&dev_a, N*sizeof(int));
  hipMalloc(&dev_b, N*sizeof(int));

  for (int i = 0; i < N; i++) {

    x[i] = 1.0f;
    y[i] = 2.0f;
    a[i] = 1;
    b[i] = 2;

  }

  hipMemcpy(dev_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  double b1=read_timer();

  flops<<<(N+447)/448,448>>>(N, 2.0f, dev_x, dev_y);

  double e1=read_timer();

  hipMemcpy(y, dev_y, N*sizeof(float), hipMemcpyDeviceToHost);

  double t1 = e1-b1;

  double b2=read_timer();

  flops<<<(N+447)/448,448>>>(N, 1.0f, dev_x, dev_y);

  double e2=read_timer();


  double t2 = e2-b2;

  double tf = t1 - t2;

  // 2 = flops per kernel (add and mult)
  printf("GFLOPS/s: %f \n", (2*N)/(tf*1e6));

  hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

  double b3=read_timer();
  iops<<<(N+447)/448,448>>>(N, 2, dev_a, dev_b);
  double e3=read_timer();

  hipMemcpy(b, dev_b, N*sizeof(int), hipMemcpyDeviceToHost);

  double t3=e3-b3;

  double ti = t3 - t2;
  
  // 2 = iops per kernel (add and mult)
  printf("GIOPS/s: %f \n", (2*N)/(ti*1e6));

  hipFree(dev_x);
  hipFree(dev_y);
  hipFree(dev_a);
  hipFree(dev_b);
}


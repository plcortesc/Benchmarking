
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

__global__
void bw(int n, float c, float *x, float *y){

  int j = blockIdx.x;

  if (j < n) y[j] = x[j] + c*y[j];

}

int main(void){

  float *x, *y, *dev_x, *dev_y;
  int N = 1<<20;

  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&dev_x, N*sizeof(float));
  hipMalloc(&dev_y, N*sizeof(float));

  for (int j = 0; j < N; j++) {
    x[j] = 4.0f;
    y[j] = 1.0f;

  }

  hipMemcpy(dev_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  hipEvent_t start1, stop1;
  hipEventCreate(&start1);
  hipEventCreate(&stop1);

  hipEventRecord(start1);

  bw<<<(N+447)/448,448>>>(N, 2.0f, dev_x, dev_y);

  hipEventRecord(stop1);

  hipEventSynchronize(stop1);
  float time = 0;
  hipEventElapsedTime(&time, start1, stop1);

  // N*4 number of bytes transferred r/w
  // 3 = rx + ry + wy
  printf("Bandwidth(GB/s): %f \n", N*4*3/time/1e6);
  hipFree(dev_x);
  hipFree(dev_y);

}

